
#include <hip/hip_runtime.h>
#include <iostream>

const int N = 10;

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void)
{
   int a[N], b[N], c[N];
   int *dev_a, *dev_b, *dev_c;
   for (int i = 0; i < N; ++i)
   {
       a[i] = i;
       b[i] = i * i;
   }
   hipMalloc((void**)&dev_a, N*sizeof(int));
   hipMalloc((void**)&dev_b, N*sizeof(int));
   hipMalloc((void**)&dev_c, N*sizeof(int));
   
   hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

   add<<<N, 1>>>(dev_a, dev_b, dev_c);

   hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

   for (int i=0; i < N; ++i)
   {
       std::cout << a[i] << "+" << b[i] << "=" << c[i] << std::endl;

   }
   return 0;
}
